#include "hip/hip_runtime.h"
// Copyright (c) 2016 Bulat Ziganshin <Bulat.Ziganshin@gmail.com>
// All rights reserved
// Part of https://github.com/Bulat-Ziganshin/Compression-Research

#include <stdio.h>
#include <vector>
#include <functional>
#include <stdint.h>

#include <helper_functions.h>          // helper for shared functions common to CUDA Samples
#include <hip/hip_runtime_api.h>               // helper functions for CUDA error checking and initialization
#include <hip/hip_runtime_api.h >
#include <hip/hip_runtime.h>

#include <hipcub/hipcub.hpp>

#include "../util/cuda_common.h"       // my own cuda-specific helper functions
#include "../util/wall_clock_timer.h"  // StartTimer() and GetTimer()
#include "../util/cpu_common.h"        // my own helper functions
#include "../util/libbsc.h"            // BSC common definitions
#include "../util/lz4_common.h"        // Utility functions from LZ4


// Parameters
const int defaultNumElements = 16<<20;
double MIN_BENCH_TIME = 0.5;  // mimimum seconds to run each bechmark


template <typename T>
__global__ void fill_with_random (T *d_array, uint32_t size)
{
    const uint32_t idx = (blockIdx.x * blockDim.x + threadIdx.x);
    if (idx >= size)  return;

    uint32_t rnd = idx*1234567891u;
    rnd = 29943829*rnd + 1013904223;    // https://en.wikipedia.org/wiki/Linear_congruential_generator
    rnd = 29943829*rnd + 1013904223;
    uint64_t rnd1 = rnd;
    rnd = 29943829*rnd + 1013904223;
    rnd = 29943829*rnd + 1013904223;
    d_array[idx] = T(rnd1<<32) + rnd;
}


template <typename Key>
double key_sort (int SORT_BYTES, size_t n, void *d_array0, hipEvent_t &start, hipEvent_t &stop)
{
    // Bit subrange [begin_bit, end_bit) of differentiating key bits
    int begin_bit = 0,  end_bit = SORT_BYTES*8;

    auto d_array = (Key*) d_array0;

    // Create a DoubleBuffer to wrap the pair of device pointers
    hipcub::DoubleBuffer<Key> d_keys (d_array, d_array + n);

    // Determine temporary device storage requirements
    void     *d_temp_storage = NULL;
    size_t   temp_storage_bytes = 0;
    checkCudaErrors( hipcub::DeviceRadixSort::SortKeys (d_temp_storage, temp_storage_bytes, d_keys, n, begin_bit, end_bit));

    // Allocate temporary storage
    checkCudaErrors( hipMalloc (&d_temp_storage, temp_storage_bytes));

    int numIterations = 0;
    double totalTime = 0;

    for ( ; totalTime < MIN_BENCH_TIME; numIterations++)
    {
        // Fill source buffer with random numbers
        fill_with_random<Key> <<< n/1024+1, 1024 >>> (d_array, n);
        checkCudaErrors( hipDeviceSynchronize());

        checkCudaErrors( hipEventRecord (start, nullptr));

        // Run sorting operation
        checkCudaErrors( hipcub::DeviceRadixSort::SortKeys (d_temp_storage, temp_storage_bytes, d_keys, n, begin_bit, end_bit));

        // Record time
        checkCudaErrors( hipEventRecord (stop, nullptr));
        checkCudaErrors( hipDeviceSynchronize());
        float start_stop;
        checkCudaErrors( hipEventElapsedTime (&start_stop, start, stop));
        totalTime += start_stop/1000; // converts milliseconds to seconds
    }

    // Release temporary storage
    checkCudaErrors( hipFree (d_temp_storage));

    return totalTime/numIterations;
}


template <typename Key, typename Value>
double keyval_sort (int SORT_BYTES, size_t n, void *d_array0, hipEvent_t &start, hipEvent_t &stop)
{
    // Bit subrange [begin_bit, end_bit) of differentiating key bits
    int begin_bit = 0,  end_bit = SORT_BYTES*8;

    auto d_array = (Key*) d_array0;
    auto d_value_array = (Value*) (d_array + 2*n);

    // Create a DoubleBuffer to wrap the pair of device pointers
    hipcub::DoubleBuffer<Key> d_keys (d_array, d_array + n);
    hipcub::DoubleBuffer<Value> d_values (d_value_array, d_value_array + n);

    // Determine temporary device storage requirements
    void     *d_temp_storage = NULL;
    size_t   temp_storage_bytes = 0;
    checkCudaErrors( hipcub::DeviceRadixSort::SortPairs (d_temp_storage, temp_storage_bytes, d_keys, d_values, n, begin_bit, end_bit));

    // Allocate temporary storage
    checkCudaErrors( hipMalloc (&d_temp_storage, temp_storage_bytes));

    int numIterations = 0;
    double totalTime = 0;

    for ( ; totalTime < MIN_BENCH_TIME; numIterations++)
    {
        // Fill source buffer with random numbers
        fill_with_random<Key> <<< n/1024+1, 1024 >>> (d_array, n);
        checkCudaErrors( hipDeviceSynchronize());

        checkCudaErrors( hipEventRecord (start, nullptr));

        // Run sorting operation
        checkCudaErrors( hipcub::DeviceRadixSort::SortPairs (d_temp_storage, temp_storage_bytes, d_keys, d_values, n, begin_bit, end_bit));

        // Record time
        checkCudaErrors( hipEventRecord (stop, nullptr));
        checkCudaErrors( hipDeviceSynchronize());
        float start_stop;
        checkCudaErrors( hipEventElapsedTime (&start_stop, start, stop));
        totalTime += start_stop/1000; // converts milliseconds to seconds
    }

    // Release temporary storage
    checkCudaErrors( hipFree (d_temp_storage));

    return totalTime/numIterations;
}


int main (int argc, char **argv)
{
    bool full = false;
    int numElements = defaultNumElements;

    while (*++argv) {
      ParseBool (*argv, "full", "", &full) ||
      ParseInt  (*argv, "",         &numElements) ||
      (printf ("radix_sort: benchmark CUB Radix Sort with various parameters.  Part of https://github.com/Bulat-Ziganshin/Compression-Research\n"
               "Usage: radix_sort [N] [full]\n"
               "  where N is the number [of millions] of elements to test\n"
               "        \"full\" enables all benchmarks\n"
              ),
       exit(1), 1);
    }

    if (numElements < 16384)
        numElements <<= 20;

    DisplayCudaDevice();

    void* d_array;
    checkCudaErrors( hipMalloc(&d_array, 4*numElements*sizeof(uint64_t)));

    hipEvent_t start, stop;
    checkCudaErrors( hipEventCreate(&start));
    checkCudaErrors( hipEventCreate(&stop));

    auto print = [&] (int bytes, int keysize, int valsize, double totalTime) {
        printf("%d/%d+%d: Throughput =%9.3lf MElements/s, Time = %.3lf ms\n",
               bytes, keysize, valsize, 1e-6 * numElements / totalTime, totalTime*1000);
    };

    printf("Sorting %dM elements:\n", numElements>>20);
    if (full)   {for(int i=1;i<=1;i++)  print (i, 1, 0, key_sort <uint8_t>  (i, numElements, d_array, start, stop));  printf("\n");}
    if (full)   {for(int i=1;i<=2;i++)  print (i, 2, 0, key_sort <uint16_t> (i, numElements, d_array, start, stop));  printf("\n");}
                {for(int i=1;i<=4;i++)  print (i, 4, 0, key_sort <uint32_t> (i, numElements, d_array, start, stop));  printf("\n");}
                {for(int i=1;i<=8;i++)  print (i, 8, 0, key_sort <uint64_t> (i, numElements, d_array, start, stop));  printf("\n");}

    if (full)   {for(int i=1;i<=1;i++)  print (i, 1, 1, keyval_sort <uint8_t,uint8_t>  (i, numElements, d_array, start, stop));  printf("\n");}
    if (full)   {for(int i=1;i<=1;i++)  print (i, 1, 2, keyval_sort <uint8_t,uint16_t> (i, numElements, d_array, start, stop));  printf("\n");}
    if (full)   {for(int i=1;i<=1;i++)  print (i, 1, 4, keyval_sort <uint8_t,uint32_t> (i, numElements, d_array, start, stop));  printf("\n");}
    if (full)   {for(int i=1;i<=1;i++)  print (i, 1, 8, keyval_sort <uint8_t,uint64_t> (i, numElements, d_array, start, stop));  printf("\n");}

    if (full)   {for(int i=1;i<=2;i++)  print (i, 2, 1, keyval_sort <uint16_t,uint8_t>  (i, numElements, d_array, start, stop));  printf("\n");}
    if (full)   {for(int i=1;i<=2;i++)  print (i, 2, 2, keyval_sort <uint16_t,uint16_t> (i, numElements, d_array, start, stop));  printf("\n");}
    if (full)   {for(int i=1;i<=2;i++)  print (i, 2, 4, keyval_sort <uint16_t,uint32_t> (i, numElements, d_array, start, stop));  printf("\n");}
    if (full)   {for(int i=1;i<=2;i++)  print (i, 2, 8, keyval_sort <uint16_t,uint64_t> (i, numElements, d_array, start, stop));  printf("\n");}

    if (full)   {for(int i=1;i<=4;i++)  print (i, 4, 1, keyval_sort <uint32_t,uint8_t>  (i, numElements, d_array, start, stop));  printf("\n");}
    if (full)   {for(int i=1;i<=4;i++)  print (i, 4, 2, keyval_sort <uint32_t,uint16_t> (i, numElements, d_array, start, stop));  printf("\n");}
                {for(int i=1;i<=4;i++)  print (i, 4, 4, keyval_sort <uint32_t,uint32_t> (i, numElements, d_array, start, stop));  printf("\n");}
                {for(int i=1;i<=4;i++)  print (i, 4, 8, keyval_sort <uint32_t,uint64_t> (i, numElements, d_array, start, stop));  printf("\n");}

    if (full)   {for(int i=1;i<=8;i++)  print (i, 8, 1, keyval_sort <uint64_t,uint8_t>  (i, numElements, d_array, start, stop));  printf("\n");}
    if (full)   {for(int i=1;i<=8;i++)  print (i, 8, 2, keyval_sort <uint64_t,uint16_t> (i, numElements, d_array, start, stop));  printf("\n");}
                {for(int i=1;i<=8;i++)  print (i, 8, 4, keyval_sort <uint64_t,uint32_t> (i, numElements, d_array, start, stop));  printf("\n");}
                {for(int i=1;i<=8;i++)  print (i, 8, 8, keyval_sort <uint64_t,uint64_t> (i, numElements, d_array, start, stop));}
    return 0;
}
