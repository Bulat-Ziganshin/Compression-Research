
#include <hip/hip_runtime.h>
/*-----------------------------------------------------------*/
/* Block Sorting, Lossless Data Compression Library.         */
/* Sort Transform (GPU version)                              */
/*-----------------------------------------------------------*/

/*--

This file is a part of bsc and/or libbsc, a program and a library for
lossless, block-sorting data compression.

Copyright (c) 2009-2011 Ilya Grebnov <ilya.grebnov@gmail.com>

See file AUTHORS for a full list of contributors.

The bsc and libbsc is free software; you can redistribute it and/or modify
it under the terms of the GNU Lesser General Public License as published by
the Free Software Foundation; either version 3 of the License, or (at your
option) any later version.

The bsc and libbsc is distributed in the hope that it will be useful, but
WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
or FITNESS FOR A PARTICULAR PURPOSE. See the GNU Lesser General Public
License for more details.

You should have received a copy of the GNU Lesser General Public License
along with the bsc and libbsc. If not, see http://www.gnu.org/licenses/.

Please see the files COPYING and COPYING.LIB for full copyright information.

See also the bsc and libbsc web site:
  http://libbsc.com/ for more information.

--*/

/*--

Sort Transform is patented by Michael Schindler under US patent 6,199,064.
However for research purposes this algorithm is included in this software.
So if you are of the type who should worry about this (making money) worry away.
The author shall have no liability with respect to the infringement of
copyrights, trade secrets or any patents by this software. In no event will
the author be liable for any lost revenue or profits or other special,
indirect and consequential damages.

Sort Transform is disabled by default and can be enabled by defining the
preprocessor macro LIBBSC_SORT_TRANSFORM_SUPPORT at compile time.

--*/

#if defined(LIBBSC_SORT_TRANSFORM_SUPPORT) && defined(LIBBSC_CUDA_SUPPORT)

#if defined(_MSC_VER)
  #pragma warning(disable : 4267)
#endif

#include <stdlib.h>
#include <memory.h>

#include "st.cuh"

#include "../libbsc.h"
#include "../platform/platform.h"

#include <cuda_runtime_api.h>
#include <device_functions.h>

#include <b40c/util/ping_pong_storage.cuh>
#include <b40c/radix_sort/enactor.cuh>

#ifdef LIBBSC_OPENMP

omp_lock_t cuda_lock;
int bsc_st_cuda_init(int features)
{
    omp_init_lock(&cuda_lock);
    return LIBBSC_NO_ERROR;
}

#else

int bsc_st_cuda_init(int features)
{
    return LIBBSC_NO_ERROR;
}

#endif

#ifndef __CUDA_ARCH__
  #define CUDA_DEVICE_ARCH              0
#else
  #define CUDA_DEVICE_ARCH              __CUDA_ARCH__
#endif

#define CUDA_DEVICE_PADDING             256
#define CUDA_NUM_THREADS_IN_BLOCK       192

#define CUDA_CTA_OCCUPANCY_SM20         8
#define CUDA_CTA_OCCUPANCY_SM12         5
#define CUDA_CTA_OCCUPANCY_SM10         4
#define CUDA_CTA_OCCUPANCY(v)           (v >= 200 ? CUDA_CTA_OCCUPANCY_SM20 : v >= 120 ? CUDA_CTA_OCCUPANCY_SM12 : CUDA_CTA_OCCUPANCY_SM10)

__global__ __launch_bounds__(CUDA_NUM_THREADS_IN_BLOCK, CUDA_CTA_OCCUPANCY(CUDA_DEVICE_ARCH))
void bsc_st567_encode_cuda_presort(unsigned char * T_device, unsigned long long * K_device, int n)
{
    __shared__ unsigned int staging[1 + CUDA_NUM_THREADS_IN_BLOCK + 7];

    unsigned int * thread_staging = &staging[threadIdx.x];
    for (int grid_size = gridDim.x * CUDA_NUM_THREADS_IN_BLOCK, block_start = blockIdx.x * CUDA_NUM_THREADS_IN_BLOCK; block_start < n; block_start += grid_size)
    {
        int index = block_start + threadIdx.x;

        {
                                  thread_staging[1                            ] = T_device[index                            ];
            if (threadIdx.x < 7 ) thread_staging[1 + CUDA_NUM_THREADS_IN_BLOCK] = T_device[index + CUDA_NUM_THREADS_IN_BLOCK]; else
            if (threadIdx.x == 7) thread_staging[-7                           ] = T_device[index - 8                        ];

            syncthreads();
        }

        {
            #if CUDA_DEVICE_ARCH >= 200
                unsigned int lo = __byte_perm(thread_staging[4], thread_staging[5], 0x0411) | __byte_perm(thread_staging[6], thread_staging[7], 0x1104);
                unsigned int hi = __byte_perm(thread_staging[0], thread_staging[1], 0x0411) | __byte_perm(thread_staging[2], thread_staging[3], 0x1104);
            #else
                unsigned int lo = (thread_staging[4] << 24) | (thread_staging[5] << 16) | (thread_staging[6] << 8) | thread_staging[7];
                unsigned int hi = (thread_staging[0] << 24) | (thread_staging[1] << 16) | (thread_staging[2] << 8) | thread_staging[3];
            #endif

            K_device[index] = (((unsigned long long)hi) << 32) | ((unsigned long long)lo);

            syncthreads();
        }
    }
}

__global__ __launch_bounds__(CUDA_NUM_THREADS_IN_BLOCK, CUDA_CTA_OCCUPANCY(CUDA_DEVICE_ARCH))
void bsc_st8_encode_cuda_presort(unsigned char * T_device, unsigned long long * K_device, unsigned char * V_device, int n)
{
    __shared__ unsigned int staging[1 + CUDA_NUM_THREADS_IN_BLOCK + 8];

    unsigned int * thread_staging = &staging[threadIdx.x];
    for (int grid_size = gridDim.x * CUDA_NUM_THREADS_IN_BLOCK, block_start = blockIdx.x * CUDA_NUM_THREADS_IN_BLOCK; block_start < n; block_start += grid_size)
    {
        int index = block_start + threadIdx.x;

        {
                                  thread_staging[1                            ] = T_device[index                            ];
            if (threadIdx.x < 8 ) thread_staging[1 + CUDA_NUM_THREADS_IN_BLOCK] = T_device[index + CUDA_NUM_THREADS_IN_BLOCK]; else
            if (threadIdx.x == 8) thread_staging[-8                           ] = T_device[index - 9                        ];

            syncthreads();
        }

        {
            #if CUDA_DEVICE_ARCH >= 200
                unsigned int lo = __byte_perm(thread_staging[5], thread_staging[6], 0x0411) | __byte_perm(thread_staging[7], thread_staging[8], 0x1104);
                unsigned int hi = __byte_perm(thread_staging[1], thread_staging[2], 0x0411) | __byte_perm(thread_staging[3], thread_staging[4], 0x1104);
            #else
                unsigned int lo = (thread_staging[5] << 24) | (thread_staging[6] << 16) | (thread_staging[7] << 8) | thread_staging[8];
                unsigned int hi = (thread_staging[1] << 24) | (thread_staging[2] << 16) | (thread_staging[3] << 8) | thread_staging[4];
            #endif

            K_device[index] = (((unsigned long long)hi) << 32) | ((unsigned long long)lo); V_device[index] = thread_staging[0];

            syncthreads();
        }
    }
}

__global__ __launch_bounds__(CUDA_NUM_THREADS_IN_BLOCK, CUDA_CTA_OCCUPANCY(CUDA_DEVICE_ARCH))
void bsc_st567_encode_cuda_postsort(unsigned char * T_device, unsigned long long * K_device, int n, unsigned long long lookup, int * I_device)
{
    int min_index = n;
    for (int grid_size = gridDim.x * CUDA_NUM_THREADS_IN_BLOCK, block_start = blockIdx.x * CUDA_NUM_THREADS_IN_BLOCK; block_start < n; block_start += grid_size)
    {
        int index = block_start + threadIdx.x;
        {
            unsigned long long value = K_device[index];
            {
                if (value == lookup && index < min_index) min_index = index;
                T_device[index] = (unsigned char)(value >> 56);
            }
        }
    }

    if (min_index != n) atomicMin(I_device, min_index);
}

__global__ __launch_bounds__(CUDA_NUM_THREADS_IN_BLOCK, CUDA_CTA_OCCUPANCY(CUDA_DEVICE_ARCH))
void bsc_st8_encode_cuda_postsort(unsigned long long * K_device, int n, unsigned long long lookup, int * I_device)
{
    int min_index = n;
    for (int grid_size = gridDim.x * CUDA_NUM_THREADS_IN_BLOCK, block_start = blockIdx.x * CUDA_NUM_THREADS_IN_BLOCK; block_start < n; block_start += grid_size)
    {
        int index = block_start + threadIdx.x;
        {
            if (K_device[index] == lookup && index < min_index) min_index = index;
        }
    }

    if (min_index != n) atomicMin(I_device, min_index);
}

int bsc_st567_encode_cuda(unsigned char * T, unsigned char * T_device, int n, int num_blocks, int k)
{
    #ifdef LIBBSC_OPENMP
        omp_set_lock(&cuda_lock);
    #endif

    int index = LIBBSC_GPU_NOT_ENOUGH_MEMORY;
    {
        unsigned long long * K_device = NULL;
        if (cudaMalloc((void **)&K_device, (n + CUDA_DEVICE_PADDING) * sizeof(unsigned long long)) == cudaSuccess)
        {
            index = LIBBSC_GPU_ERROR;

            bsc_st567_encode_cuda_presort<<<num_blocks, CUDA_NUM_THREADS_IN_BLOCK>>>(T_device, K_device, n);

            cudaError_t status = cudaSuccess;
            {
                b40c::util::PingPongStorage<unsigned long long> storage(K_device);

                b40c::radix_sort::Enactor enactor;
                if (k == 5) status = enactor.Sort<16, 40, b40c::radix_sort::LARGE_SIZE>(storage, n);
                if (k == 6) status = enactor.Sort< 8, 48, b40c::radix_sort::LARGE_SIZE>(storage, n);
                if (k == 7) status = enactor.Sort< 0, 56, b40c::radix_sort::LARGE_SIZE>(storage, n);

                if (status == cudaErrorMemoryAllocation) index = LIBBSC_GPU_NOT_ENOUGH_MEMORY;

                if (status == cudaSuccess && storage.selector == 1)
                {
                    cudaMemcpy(K_device, storage.d_keys[1], n * sizeof(unsigned long long), cudaMemcpyDeviceToDevice);
                }

                if (storage.d_keys[1] != NULL) cudaFree(storage.d_keys[1]);
            }

            if (status == cudaSuccess)
            {
                unsigned long long lookup;
                {
                    unsigned int lo = (T[3    ] << 24) | (T[4] << 16) | (T[5] << 8) | T[6];
                    unsigned int hi = (T[n - 1] << 24) | (T[0] << 16) | (T[1] << 8) | T[2];

                    lookup = (((unsigned long long)hi) << 32) | ((unsigned long long)lo);

                    cudaMemcpy(T_device - sizeof(int), &n, sizeof(int), cudaMemcpyHostToDevice);
                }

                bsc_st567_encode_cuda_postsort<<<num_blocks, CUDA_NUM_THREADS_IN_BLOCK>>>(T_device, K_device, n, lookup, (int *)(T_device - sizeof(int)));

                cudaFree(K_device);

                #ifdef LIBBSC_OPENMP
                    omp_unset_lock(&cuda_lock);
                #endif

                cudaMemcpy(T_device + n, T_device - sizeof(int), sizeof(int), cudaMemcpyDeviceToDevice);
                cudaMemcpy(T, T_device, n + sizeof(int), cudaMemcpyDeviceToHost);

                if (cudaGetLastError() == cudaSuccess)
                {
                    index = *(int *)(T + n);
                }

                return index;
            }
            cudaFree(K_device);
        }
    }

    #ifdef LIBBSC_OPENMP
        omp_unset_lock(&cuda_lock);
    #endif

    return index;
}

int bsc_st8_encode_cuda(unsigned char * T, unsigned char * T_device, int n, int num_blocks)
{
    #ifdef LIBBSC_OPENMP
        omp_set_lock(&cuda_lock);
    #endif

    int index = LIBBSC_GPU_NOT_ENOUGH_MEMORY;
    {
        unsigned char * V_device = NULL;
        if (cudaMalloc((void **)&V_device, (n + CUDA_DEVICE_PADDING) * sizeof(unsigned char)) == cudaSuccess)
        {
            unsigned long long * K_device = NULL;
            if (cudaMalloc((void **)&K_device, (n + CUDA_DEVICE_PADDING) * sizeof(unsigned long long)) == cudaSuccess)
            {
                index = LIBBSC_GPU_ERROR;

                bsc_st8_encode_cuda_presort<<<num_blocks, CUDA_NUM_THREADS_IN_BLOCK>>>(T_device, K_device, V_device, n);

                cudaError_t status = cudaSuccess;
                {
                    b40c::util::PingPongStorage<unsigned long long, unsigned char> storage(K_device, V_device);

                    b40c::radix_sort::Enactor enactor;

                    status = enactor.Sort<b40c::radix_sort::LARGE_SIZE>(storage, n);

                    if (status == cudaErrorMemoryAllocation) index = LIBBSC_GPU_NOT_ENOUGH_MEMORY;

                    if (status == cudaSuccess && storage.selector == 1)
                    {
                        cudaMemcpy(K_device, storage.d_keys[1]  , n * sizeof(unsigned long long), cudaMemcpyDeviceToDevice);
                        cudaMemcpy(V_device, storage.d_values[1], n * sizeof(unsigned char     ), cudaMemcpyDeviceToDevice);
                    }

                    if (storage.d_keys[1]   != NULL) cudaFree(storage.d_keys[1]  );
                    if (storage.d_values[1] != NULL) cudaFree(storage.d_values[1]);
                }

                if (status == cudaSuccess)
                {
                    unsigned long long lookup;
                    {
                        unsigned int lo = (T[4] << 24) | (T[5] << 16) | (T[6] << 8) | T[7];
                        unsigned int hi = (T[0] << 24) | (T[1] << 16) | (T[2] << 8) | T[3];

                        lookup = (((unsigned long long)hi) << 32) | ((unsigned long long)lo);

                        cudaMemcpy(V_device + n, &n, sizeof(int), cudaMemcpyHostToDevice);
                    }

                    bsc_st8_encode_cuda_postsort<<<num_blocks, CUDA_NUM_THREADS_IN_BLOCK>>>(K_device, n, lookup, (int *)(V_device + n));

                    cudaFree(K_device);

                    #ifdef LIBBSC_OPENMP
                        omp_unset_lock(&cuda_lock);
                    #endif

                    cudaMemcpy(T, V_device, n + sizeof(int), cudaMemcpyDeviceToHost);

                    cudaFree(V_device);

                    if (cudaGetLastError() == cudaSuccess)
                    {
                        index = *(int *)(T + n);
                    }

                    return index;
                }
                cudaFree(K_device);
            }
            cudaFree(V_device);
        }
    }

    #ifdef LIBBSC_OPENMP
        omp_unset_lock(&cuda_lock);
    #endif

    return index;
}

int bsc_st_encode_cuda(unsigned char * T, int n, int k, int features)
{
    if ((T == NULL) || (n < 0)) return LIBBSC_BAD_PARAMETER;
    if ((k < 5) || (k > 8))     return LIBBSC_BAD_PARAMETER;
    if (n <= 1)                 return 0;

    int num_blocks = 1;
    {
        cudaDeviceProp deviceProperties;
        {
            int deviceId; if (cudaGetDevice(&deviceId) != cudaSuccess || cudaGetDeviceProperties(&deviceProperties, deviceId) != cudaSuccess)
            {
                return LIBBSC_GPU_NOT_SUPPORTED;
            }
        }

        if (deviceProperties.major * 10 + deviceProperties.minor <= 10) return LIBBSC_GPU_NOT_SUPPORTED;
        num_blocks = CUDA_CTA_OCCUPANCY(deviceProperties.major * 100 + deviceProperties.minor * 10) * deviceProperties.multiProcessorCount;

        if (num_blocks > ((n + CUDA_NUM_THREADS_IN_BLOCK - 1) / CUDA_NUM_THREADS_IN_BLOCK)) num_blocks = (n + CUDA_NUM_THREADS_IN_BLOCK - 1) / CUDA_NUM_THREADS_IN_BLOCK;
        if (num_blocks <= 0) num_blocks = 1;
    }

    int index = LIBBSC_GPU_NOT_ENOUGH_MEMORY;
    {
        unsigned char * T_device = NULL;
        if (cudaMalloc((void **)&T_device, n + 2 * CUDA_DEVICE_PADDING) == cudaSuccess)
        {
            cudaMemcpy(T_device + CUDA_DEVICE_PADDING    , T                             , n                  , cudaMemcpyHostToDevice  );
            cudaMemcpy(T_device + CUDA_DEVICE_PADDING + n, T_device + CUDA_DEVICE_PADDING, CUDA_DEVICE_PADDING, cudaMemcpyDeviceToDevice);
            cudaMemcpy(T_device                          , T_device + n                  , CUDA_DEVICE_PADDING, cudaMemcpyDeviceToDevice);

            if (k >= 5 && k <= 7) index = bsc_st567_encode_cuda(T, T_device + CUDA_DEVICE_PADDING, n, num_blocks, k);
            if (k == 8)           index = bsc_st8_encode_cuda  (T, T_device + CUDA_DEVICE_PADDING, n, num_blocks   );

            cudaFree(T_device);
        }
    }

    return index;
}

#endif

/*-----------------------------------------------------------*/
/* End                                                 st.cu */
/*-----------------------------------------------------------*/
