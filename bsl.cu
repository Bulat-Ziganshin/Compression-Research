#include "hip/hip_runtime.h"
// Copyright (c) 2016 Bulat Ziganshin <Bulat.Ziganshin@gmail.com>
// All rights reserved
// Part of https://github.com/Bulat-Ziganshin/Compression-Research

#include <stdio.h>
#include <vector>
#include <functional>

#include <helper_functions.h>  // helper for shared functions common to CUDA Samples
#include <hip/hip_runtime_api.h>       // helper functions for CUDA error checking and initialization
#include <hip/hip_runtime_api.h >
#include <hip/hip_runtime.h>

#include "lib/wall_clock_timer.h"  // StartTimer() and GetTimer()
#include "lib/cpu_common.h"        // my own helper functions
#include "lib/cuda_common.h"       // my own cuda-specific helper functions
#include "lib/libbsc.h"            // BSC common definitions

const int WARP_SIZE = 32;
typedef unsigned char byte;

// Parameters
const int DEFAULT_BUFSIZE = 128*1024*1024;
const int CHUNK = 4*1024;
#define SYNC_WARP __threadfence_block  /* alternatively, __syncthreads or, better, __threadfence_warp */

#include "lzp/lzp-cpu-bsc.cpp"
#include "lzp/lzp-cpu-bsc-mod.cpp"
#include "lzp/lzp-cpu-rollhash.cpp"

#include "bwt/sais.c"              // OpenBWT implementation
#define LIBBSC_SORT_TRANSFORM_SUPPORT
#include "st/st.cpp"               // BSC Sort Transform implementation

#include "mtf/qlfc-cpu.cpp"
#include "mtf/mtf_scalar.cu"
#include "mtf/mtf_2symbols.cu"
#include "mtf/mtf_2buffers.cu"
#include "mtf/mtf_2buffers_depth32.cu"
#include "mtf/mtf_4by8.cu"
#include "mtf/mtf_thread.cu"
#include "mtf/mtf_thread_by4.cu"

// In-place RLE transformation (run lengths are dropped!)
int rle (byte* buf, int size)
{
    int c = -1,  run = 0;
    auto out = buf;
    for (size_t i = 0; i < size; i++)
    {
        buf[i]==c?  run++  :  (run=1, c = *out++ = buf[i]);
    }
    return out-buf;
}

int main (int argc, char **argv)
{
    bool display_gpu = true;
    bool apply_lzp = true;
    bool apply_bwt = true;
    bool apply_rle = true;
    bool apply_mtf = true;
    enum STAGE {LZP, BWT, MTF};  const int STAGES = 3;
    int snum[] = {-1,-1,-1},  lzpHashSize = 15,  lzpMinLen = 32;
    size_t bufsize = DEFAULT_BUFSIZE;
    char *comment;
    int error = 0;

    auto src_argv = argv,  dst_argv = argv;
    while (*++src_argv) {
      ParseBool (*src_argv, "-gpu", "-nogpu", &display_gpu) ||
      ParseBool (*src_argv, "-lzp", "-nolzp", &apply_lzp) ||
      ParseBool (*src_argv, "-bwt", "-nobwt", &apply_bwt) ||
      ParseBool (*src_argv, "-rle", "-norle", &apply_rle) ||
      ParseBool (*src_argv, "-mtf", "-nomtf", &apply_mtf) ||
      ParseInt  (*src_argv, "-lzp",           &snum[LZP]) ||
      ParseInt  (*src_argv, "-bwt",           &snum[BWT]) ||
      ParseInt  (*src_argv, "-mtf",           &snum[MTF]) ||
      ParseInt  (*src_argv, "-b",             &bufsize) ||
      ParseInt  (*src_argv, "-h",             &lzpHashSize) ||
      ParseInt  (*src_argv, "-l",             &lzpMinLen) ||
      ParseStr  (*src_argv, "-rem",           &comment) ||
      UnknownOption (*src_argv, &error) ||
      (*++dst_argv = *src_argv);
    }
    *++dst_argv = 0;  argc = dst_argv - argv;

    if (bufsize < 100*1000)
        bufsize <<= 20;  // megabytes

    if (!(argc==2 || argc==3) || error) {
        printf ("BSL: the block-sorting lab.  Part of https://github.com/Bulat-Ziganshin/Compression-Research\n"
                "Usage: mtf [options] infile [outfile]\n"
                "  -nogpu   skip GPU name output\n"
                "  -nolzp   skip LZP transformation\n"
                "  -nobwt   skip BWT transformation\n"
                "  -norle   skip RLE transformation\n"
                "  -nomtf   skip MTF transformation\n"
                "  -bN      buffer N (mebi)bytes (default %d MiB)\n"
                "  -lzpN    perform only LZP transformation number N\n"
                "  -hN      set LZP hash size (default 2^%d hash entries)\n"
                "  -lN      set LZP minLen (default %d)\n"
                "  -bwtN    perform only sorting transformation number N\n"
                "  -mtfN    perform only MTF transformation number N\n"
                "  -rem...  ignored by the program\n"
                , lzpHashSize, lzpMinLen, DEFAULT_BUFSIZE>>20);
        return argc==1 && !error?  0 : 1;
    }

    unsigned char* d_inbuf;
    unsigned char* d_outbuf;
    checkCudaErrors( hipMalloc((void**)(&d_inbuf),  bufsize+CHUNK*2+256));  // up to CHUNK*2 extra bytes may be processed, plus a few extra bytes may be read after bufend
    checkCudaErrors( hipMalloc((void**)(&d_outbuf), bufsize+CHUNK*2+256));

    hipEvent_t start, stop;
    checkCudaErrors( hipEventCreate(&start));
    checkCudaErrors( hipEventCreate(&stop));

    unsigned char* inbuf  = new unsigned char[bufsize];
    unsigned char* outbuf = new unsigned char[bufsize];
    int*      bwt_tempbuf = apply_bwt? new int[bufsize] : 0;

    double insize = 0,  after_lzp = 0,  outsize = 0;
    double size[STAGES][100] = {0},  duration[STAGES][100] = {0};  char *name[STAGES][100] = {{},{},{"cpu (1 thread)"}};

    int inbytes, num, stage, bsc_errcode;
    auto cpu_time_run = [&] (char *_name, std::function<int(void)> stage_f) {
        name[stage][num] = _name;
        if (num == snum[stage]  ||  snum[stage] < 0)
        {
            StartTimer();
            bsc_errcode  =  stage_f();
            duration[stage][num] += GetTimer();

            if (bsc_errcode < 0  &&  bsc_errcode != LIBBSC_NOT_COMPRESSIBLE) {
                printf ("%s failed with errcode %d\n", name, bsc_errcode);
                exit(4);
            }
            size[stage][num]  +=  (bsc_errcode != LIBBSC_NOT_COMPRESSIBLE?  bsc_errcode : inbytes);
        }
        num++;
    };


    FILE* infile  = fopen (argv[1], "rb");
    FILE* outfile = fopen (argv[2]? argv[2] : "nul", "wb");
    if (!infile) {
        printf ("Can't open infile %s\n", argv[1]);
        return 2;
    }
    if (!outfile) {
        printf ("Can't open outfile %s\n", argv[3]);
        return 3;
    }
    if (display_gpu)
        DisplayCudaDevice();


    // All preparations now are done. Now we are in the Analysis stage, processing input data with various algos and recording speed/outsize of every experiment
    while (!!(inbytes = fread(inbuf,1,bufsize,infile)))
    {
        insize += inbytes;
        byte *ptr = inbuf;  size_t outbytes = inbytes;  // output buffer

        if (apply_lzp) {
            lzp_cpu_bsc (inbuf, inbuf+inbytes, outbuf, outbuf+inbytes, lzpHashSize, lzpMinLen);   // "massage" data in order to provide equal conditions for the both following lzp routines

            num = 1,  stage = LZP;
            cpu_time_run ("lzp_cpu_bsc     ", [&] {return lzp_cpu_bsc      (inbuf, inbuf+inbytes, outbuf, outbuf+inbytes, lzpHashSize, lzpMinLen);});
            cpu_time_run ("lzp_cpu_bsc_mod ", [&] {return lzp_cpu_bsc_mod  (inbuf, inbuf+inbytes, outbuf, outbuf+inbytes, lzpHashSize, lzpMinLen);});
            cpu_time_run ("lzp_cpu_rollhash", [&] {return lzp_cpu_rollhash (inbuf, inbuf+inbytes, outbuf, outbuf+inbytes, lzpHashSize, lzpMinLen);});

            if (bsc_errcode != LIBBSC_NOT_COMPRESSIBLE)
                memcpy (inbuf, outbuf, inbytes=bsc_errcode);
        }
        after_lzp += inbytes;

        if (apply_bwt) {
            StartTimer();
            auto bwt_errcode  =  sais_bwt (inbuf, outbuf, bwt_tempbuf, inbytes);
            duration[BWT][0] += GetTimer();
            if (bwt_errcode < 0) {
                printf ("BWT failed with errcode %d\n", bwt_errcode);
                return 5;
            }
            memcpy (inbuf, outbuf, inbytes);
        }

        if (0 == snum[MTF]  ||  snum[MTF] < 0) {
            StartTimer();
                unsigned char MTFTable[ALPHABET_SIZE];
                ptr = qlfc (inbuf, outbuf, inbytes, MTFTable);
                outbytes = outbuf+inbytes - ptr;
            duration[MTF][0] += GetTimer();
        }
        int num = 1;

        if (apply_rle) {
            inbytes = rle(inbuf,inbytes);
        }

        checkCudaErrors( hipMemcpy (d_inbuf, inbuf, inbytes, hipMemcpyHostToDevice));
        checkCudaErrors( hipDeviceSynchronize());

        auto time_run = [&] (char *_name, std::function<void(void)> f) {
            name[MTF][num] = _name;
            if (num == snum[MTF]  ||  snum[MTF] < 0)
            {
                checkCudaErrors( hipEventRecord (start, nullptr));
                f();
                checkCudaErrors( hipEventRecord (stop, nullptr));
                checkCudaErrors( hipDeviceSynchronize());

                if (num == snum[MTF]) {
                    checkCudaErrors( hipMemcpy (outbuf, d_outbuf, inbytes, hipMemcpyDeviceToHost));
                    checkCudaErrors( hipDeviceSynchronize());
                    ptr = outbuf;
                    outbytes = inbytes;
                }

                float start_stop;
                checkCudaErrors( hipEventElapsedTime (&start_stop, start, stop));
                duration[MTF][num] += start_stop;
            }
            num++;
        };

{
        const int NUM_WARPS = 4;
        time_run ("mtf_scalar        ", [&] {mtf_scalar    <CHUNK,NUM_WARPS>       <<<(inbytes-1)/(CHUNK*NUM_WARPS)+1,   NUM_WARPS*WARP_SIZE>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_2symbols      ", [&] {mtf_2symbols  <CHUNK,NUM_WARPS>       <<<(inbytes-1)/(CHUNK*NUM_WARPS)+1,   NUM_WARPS*WARP_SIZE>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_2buffers      ", [&] {mtf_2buffers  <CHUNK,NUM_WARPS>       <<<(inbytes-1)/(CHUNK*NUM_WARPS*2)+1, NUM_WARPS*WARP_SIZE>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_2buffers<32>  ", [&] {mtf_2buffers_depth32 <CHUNK,NUM_WARPS><<<(inbytes-1)/(CHUNK*NUM_WARPS*2)+1, NUM_WARPS*WARP_SIZE>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_3buffers<32>  ", [&] {mtf_2buffers_depth32 <CHUNK,NUM_WARPS,3><<<(inbytes-1)/(CHUNK*NUM_WARPS*3)+1, NUM_WARPS*WARP_SIZE>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_4buffers<32>  ", [&] {mtf_2buffers_depth32 <CHUNK,NUM_WARPS,4><<<(inbytes-1)/(CHUNK*NUM_WARPS*4)+1, NUM_WARPS*WARP_SIZE>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
}
{
        const int NUM_THREADS = 8*WARP_SIZE,  NUM_BUFFERS = NUM_THREADS/8;
        time_run ("mtf_4by8          ", [&]{mtf_4by8<CHUNK,NUM_THREADS,NUM_BUFFERS><<<(inbytes-1)/(CHUNK*NUM_BUFFERS)+1,         NUM_THREADS>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
}
{
        const int NUM_THREADS = 4*WARP_SIZE,  NUM_BUFFERS = NUM_THREADS/4;
        time_run ("mtf_8by4          ", [&]{mtf_4by8<CHUNK,NUM_THREADS,NUM_BUFFERS><<<(inbytes-1)/(CHUNK*NUM_BUFFERS)+1,         NUM_THREADS>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
}
        time_run ("mtf_thread        ", [&] {mtf_thread    <CHUNK>                 <<<(inbytes-1)/(CHUNK*WARP_SIZE)+1,             WARP_SIZE>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_thread_by4    ", [&] {mtf_thread_by4<CHUNK>                 <<<(inbytes-1)/(CHUNK*WARP_SIZE)+1,             WARP_SIZE>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});

        const int NUM_THREADS = 1*WARP_SIZE;
        time_run ("mtf_thread<8>     ", [&] {mtf_thread    <CHUNK,NUM_THREADS,8>   <<<(inbytes-1)/(CHUNK*NUM_THREADS)+1,         NUM_THREADS>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_thread<16>    ", [&] {mtf_thread    <CHUNK,NUM_THREADS,16>  <<<(inbytes-1)/(CHUNK*NUM_THREADS)+1,         NUM_THREADS>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_thread<32>    ", [&] {mtf_thread    <CHUNK,NUM_THREADS,32>  <<<(inbytes-1)/(CHUNK*NUM_THREADS)+1,         NUM_THREADS>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_thread<64>    ", [&] {mtf_thread    <CHUNK,NUM_THREADS,64>  <<<(inbytes-1)/(CHUNK*NUM_THREADS)+1,         NUM_THREADS>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});

        time_run ("mtf_thread_by4<8> ", [&] {mtf_thread_by4<CHUNK,NUM_THREADS,8>   <<<(inbytes-1)/(CHUNK*NUM_THREADS)+1,         NUM_THREADS>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_thread_by4<16>", [&] {mtf_thread_by4<CHUNK,NUM_THREADS,16>  <<<(inbytes-1)/(CHUNK*NUM_THREADS)+1,         NUM_THREADS>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_thread_by4<32>", [&] {mtf_thread_by4<CHUNK,NUM_THREADS,32>  <<<(inbytes-1)/(CHUNK*NUM_THREADS)+1,         NUM_THREADS>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_thread_by4<64>", [&] {mtf_thread_by4<CHUNK,NUM_THREADS,64>  <<<(inbytes-1)/(CHUNK*NUM_THREADS)+1,         NUM_THREADS>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});

        fwrite (ptr, 1, outbytes, outfile);
        outsize += outbytes;
    }


    // The Analysis stage now is finished, we are going to display the collected data in fancy way
    auto print_stage_stats = [&] (int num, char *name, double insize, double outsize, double duration) {
        if (num >= 0)
            printf("[%2d] ", num);
        printf("%s: ", name);
        if (outsize >= 0  &&  outsize != insize)
            printf("%.0lf => %.0lf (%.2lf%%)", insize, outsize, outsize*100/insize);
        if (duration) {
            auto speed = ((1000/duration) *  insize) / (1 << 20);
            int digits = speed<10?3:speed<100?2:0;
            printf("%*.*lf MiB/s,  %.3lf ms", (num>=0?5:0), digits, speed, duration);
        }
        printf("\n");
    };

    for (int i=1; i<100; i++) {
        if (duration[LZP][i]) {
            print_stage_stats (i, name[LZP][i], insize, size[LZP][i], duration[LZP][i]);
        }
    }

    if (apply_bwt)  print_stage_stats (-1, "bwt", after_lzp, -1, duration[BWT][0]);
    if (apply_rle)  print_stage_stats (-1, "rle", after_lzp, outsize, 0);

    for (int i=0; i<100; i++) {
        if (duration[MTF][i]) {
            char in_speed[100], out_speed[100];
            sprintf( in_speed,   "%5.0lf", ((1000/duration[MTF][i]) *  insize) / (1 << 20));
            sprintf(out_speed, " /%5.0lf", ((1000/duration[MTF][i]) * outsize) / (1 << 20));
            printf("[%2d] %-*s: %s%s MiB/s,  %.3lf ms\n", i, strlen(name[MTF][2]), name[MTF][i], in_speed, (outsize!=insize?out_speed:""), duration[MTF][i]);
        }
    }
    fclose(infile);
    fclose(outfile);
    hipProfilerStop();
    return 0;
}
