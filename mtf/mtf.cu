#include "hip/hip_runtime.h"
// Copyright (C) 2016 Bulat Ziganshin
// All right reserved
// Part of https://github.com/Bulat-Ziganshin/Compression-Research

#include <stdio.h>
#include <vector>
#include <functional>

#include <helper_functions.h>  // helper for shared functions common to CUDA Samples
#include <hip/hip_runtime_api.h>       // helper functions for CUDA error checking and initialization
#include <hip/hip_runtime_api.h >
#include <hip/hip_runtime.h>

#include "wall_clock_timer.h"  // StartTimer() and GetTimer()
#include "cuda_common.h"       // my own helper functions
#include "sais.c"              // OpenBWT implementation

const int ALPHABET_SIZE = 256;
const int WARP_SIZE = 32;
typedef unsigned char byte;

// Parameters
const int BUFSIZE = 128*1024*1024;
const int CHUNK = 4*1024;
#define SYNC_WARP __threadfence_block  /* alternatively, __syncthreads or, better, __threadfence_warp */

#include "qlfc-cpu.cpp"
#include "mtf_scalar.cu"
#include "mtf_2symbols.cu"
#include "mtf_2buffers.cu"
#include "mtf_2buffers_depth32.cu"
#include "mtf_4by8.cu"
#include "mtf_thread.cu"
#include "mtf_thread_by4.cu"

// In-place RLE transformation (run lengths are dropped!)
int rle (byte* buf, int size)
{
    int c = -1, run=0;
    auto out = buf;
    for (size_t i = 0; i < size; i++)
    {
        buf[i]==c?  run++  :  (run=1, c = *out++ = buf[i]);
    }
    return out-buf;
}


int main (int argc, char **argv)
{
    bool apply_gpu_display = true;
    if (argv[1] && strcmp(argv[1],"-nogpu")==0) {
        apply_gpu_display = false;
        argv++, argc--;
    }

    bool apply_bwt = true;
    if (argv[1] && strcmp(argv[1],"-nobwt")==0) {
        apply_bwt = false;
        argv++, argc--;
    }

    bool apply_rle = true;
    if (argv[1] && strcmp(argv[1],"-norle")==0) {
        apply_rle = false;
        argv++, argc--;
    }

    if (!(argc==2 || argc==4)) {
        printf ("Usage: mtf [options] infile [N outfile]\n"
                "  N is the number of function those output will be saved\n"
                "  -nogpu   skip GPU name output\n"
                "  -nobwt   skip BWT transformation\n"
                "  -norle   skip RLE transformation\n"
                );
        return 0;
    }

    unsigned char* d_inbuf;
    unsigned char* d_outbuf;
    checkCudaErrors( hipMalloc((void**)(&d_inbuf),  BUFSIZE+CHUNK*2));  // up to CHUNK*2 extra bytes may be processed
    checkCudaErrors( hipMalloc((void**)(&d_outbuf), BUFSIZE+CHUNK*2));

    hipEvent_t start, stop;
    checkCudaErrors( hipEventCreate(&start));
    checkCudaErrors( hipEventCreate(&stop));

    unsigned char* inbuf  = new unsigned char[BUFSIZE];
    unsigned char* outbuf = new unsigned char[BUFSIZE];
    int*      bwt_tempbuf = new int          [BUFSIZE];

    double insize = 0,  outsize = 0,  duration[100] = {0};  char *mtf_name[100] = {"cpu (1 thread)"};

    FILE* infile  = fopen (argv[1], "rb");
    FILE* outfile = fopen (argv[2]? argv[3] : "nul", "wb");
    if (!infile) {
        printf ("Can't open infile %s\n", argv[1]);
        return 1;
    }
    if (!outfile) {
        printf ("Can't open outfile %s\n", argv[3]);
        return 1;
    }
    int save_num  =  argc==4? atoi(argv[2]) : 0;
    if (apply_gpu_display)
        DisplayCudaDevice();


    for (int inbytes; !!(inbytes = fread(inbuf,1,BUFSIZE,infile)); )
    {
        if (apply_bwt) {
            auto bwt_errcode  =  sais_bwt (inbuf, outbuf, bwt_tempbuf, inbytes);
            if (bwt_errcode < 0) {
                printf ("BWT failed with errcode %d\n", bwt_errcode);
                return 2;
            }
            memcpy (inbuf, outbuf, inbytes);
        }

        StartTimer();
            unsigned char MTFTable[ALPHABET_SIZE];
            auto ptr = qlfc (inbuf, outbuf, inbytes, MTFTable);
            auto outbytes = outbuf+inbytes - ptr;
        duration[0] += GetTimer();
        int num = 1;

        insize += inbytes;
        if (apply_rle)
            inbytes = rle(inbuf,inbytes);

        checkCudaErrors( hipMemcpy (d_inbuf, inbuf, inbytes, hipMemcpyHostToDevice));
        checkCudaErrors( hipDeviceSynchronize());

        auto time_run = [&] (char *name, std::function<void(void)> f) {
            mtf_name[num] = name;
            checkCudaErrors( hipEventRecord (start, nullptr));
            f();
            checkCudaErrors( hipEventRecord (stop, nullptr));
            checkCudaErrors( hipDeviceSynchronize());

            if (num == save_num) {
                checkCudaErrors( hipMemcpy (outbuf, d_outbuf, inbytes, hipMemcpyDeviceToHost));
                checkCudaErrors( hipDeviceSynchronize());
                ptr = outbuf;
                outbytes = inbytes;
            }

            float start_stop;
            checkCudaErrors( hipEventElapsedTime (&start_stop, start, stop));
            duration[num] += start_stop;
            num++;
        };

{
        const int NUM_WARPS = 4;
        time_run ("mtf_scalar        ", [&] {mtf_scalar    <CHUNK,NUM_WARPS>       <<<(inbytes-1)/(CHUNK*NUM_WARPS)+1,   NUM_WARPS*WARP_SIZE>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_2symbols      ", [&] {mtf_2symbols  <CHUNK,NUM_WARPS>       <<<(inbytes-1)/(CHUNK*NUM_WARPS)+1,   NUM_WARPS*WARP_SIZE>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_2buffers      ", [&] {mtf_2buffers  <CHUNK,NUM_WARPS>       <<<(inbytes-1)/(CHUNK*NUM_WARPS*2)+1, NUM_WARPS*WARP_SIZE>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_2buffers<32>  ", [&] {mtf_2buffers_depth32 <CHUNK,NUM_WARPS><<<(inbytes-1)/(CHUNK*NUM_WARPS*2)+1, NUM_WARPS*WARP_SIZE>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_3buffers<32>  ", [&] {mtf_2buffers_depth32 <CHUNK,NUM_WARPS,3><<<(inbytes-1)/(CHUNK*NUM_WARPS*3)+1, NUM_WARPS*WARP_SIZE>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_4buffers<32>  ", [&] {mtf_2buffers_depth32 <CHUNK,NUM_WARPS,4><<<(inbytes-1)/(CHUNK*NUM_WARPS*4)+1, NUM_WARPS*WARP_SIZE>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
}
{
        const int NUM_THREADS = 8*WARP_SIZE,  NUM_BUFFERS = NUM_THREADS/8;
        time_run ("mtf_4by8          ", [&]{mtf_4by8<CHUNK,NUM_THREADS,NUM_BUFFERS><<<(inbytes-1)/(CHUNK*NUM_BUFFERS)+1,         NUM_THREADS>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
}
{
        const int NUM_THREADS = 4*WARP_SIZE,  NUM_BUFFERS = NUM_THREADS/4;
        time_run ("mtf_8by4          ", [&]{mtf_4by8<CHUNK,NUM_THREADS,NUM_BUFFERS><<<(inbytes-1)/(CHUNK*NUM_BUFFERS)+1,         NUM_THREADS>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
}
        time_run ("mtf_thread        ", [&] {mtf_thread    <CHUNK>                 <<<(inbytes-1)/(CHUNK*WARP_SIZE)+1,             WARP_SIZE>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_thread_by4    ", [&] {mtf_thread_by4<CHUNK>                 <<<(inbytes-1)/(CHUNK*WARP_SIZE)+1,             WARP_SIZE>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});

        const int NUM_THREADS = 1*WARP_SIZE;
        time_run ("mtf_thread<8>     ", [&] {mtf_thread    <CHUNK,NUM_THREADS,8>   <<<(inbytes-1)/(CHUNK*NUM_THREADS)+1,         NUM_THREADS>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_thread<16>    ", [&] {mtf_thread    <CHUNK,NUM_THREADS,16>  <<<(inbytes-1)/(CHUNK*NUM_THREADS)+1,         NUM_THREADS>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_thread<32>    ", [&] {mtf_thread    <CHUNK,NUM_THREADS,32>  <<<(inbytes-1)/(CHUNK*NUM_THREADS)+1,         NUM_THREADS>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_thread<64>    ", [&] {mtf_thread    <CHUNK,NUM_THREADS,64>  <<<(inbytes-1)/(CHUNK*NUM_THREADS)+1,         NUM_THREADS>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});

        time_run ("mtf_thread_by4<8> ", [&] {mtf_thread_by4<CHUNK,NUM_THREADS,8>   <<<(inbytes-1)/(CHUNK*NUM_THREADS)+1,         NUM_THREADS>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_thread_by4<16>", [&] {mtf_thread_by4<CHUNK,NUM_THREADS,16>  <<<(inbytes-1)/(CHUNK*NUM_THREADS)+1,         NUM_THREADS>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_thread_by4<32>", [&] {mtf_thread_by4<CHUNK,NUM_THREADS,32>  <<<(inbytes-1)/(CHUNK*NUM_THREADS)+1,         NUM_THREADS>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_thread_by4<64>", [&] {mtf_thread_by4<CHUNK,NUM_THREADS,64>  <<<(inbytes-1)/(CHUNK*NUM_THREADS)+1,         NUM_THREADS>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});

        fwrite (ptr, 1, outbytes, outfile);
        outsize += outbytes;
    }

    printf("rle: %.0lf => %.0lf (%.2lf%%)\n", insize, outsize, outsize*100.0/insize);
    for (int i=0; i<sizeof(duration)/sizeof(*duration); i++) {
        if (duration[i]) {
            char in_speed[100], out_speed[100];
            sprintf( in_speed,   "%5.0lf", ((1000/duration[i]) *  insize) / (1 << 20));
            sprintf(out_speed, " /%5.0lf", ((1000/duration[i]) * outsize) / (1 << 20));
            printf("[%2d] %-*s: %s%s MiB/s,  %.3lf ms\n", i, strlen(mtf_name[2]), mtf_name[i], in_speed, (apply_rle?out_speed:""), duration[i]);
        }
    }
    fclose(infile);
    fclose(outfile);
    hipProfilerStop();
    return 0;
}
