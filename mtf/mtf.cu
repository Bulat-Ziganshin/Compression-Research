#include "hip/hip_runtime.h"
// Copyright (C) 2016 Bulat Ziganshin
// All right reserved
// Part of https://github.com/Bulat-Ziganshin/Compression-Research

#include <stdio.h>
#include <vector>
#include <functional>

#include <helper_functions.h>  // helper for shared functions common to CUDA Samples
#include <hip/hip_runtime_api.h>       // helper functions for CUDA error checking and initialization
#include <hip/hip_runtime_api.h >
#include <hip/hip_runtime.h>

#include "wall_clock_timer.h"  // StartTimer() and GetTimer()
#include "cpu_common.h"        // my own helper functions
#include "cuda_common.h"       // my own cuda-specific helper functions
#include "sais.c"              // OpenBWT implementation

const int ALPHABET_SIZE = 256;
const int WARP_SIZE = 32;
typedef unsigned char byte;

// Parameters
const int DEFAULT_BUFSIZE = 128*1024*1024;
const int CHUNK = 4*1024;
#define SYNC_WARP __threadfence_block  /* alternatively, __syncthreads or, better, __threadfence_warp */

#include "lzp-cpu.cpp"
#include "qlfc-cpu.cpp"
#include "mtf_scalar.cu"
#include "mtf_2symbols.cu"
#include "mtf_2buffers.cu"
#include "mtf_2buffers_depth32.cu"
#include "mtf_4by8.cu"
#include "mtf_thread.cu"
#include "mtf_thread_by4.cu"

// In-place RLE transformation (run lengths are dropped!)
int rle (byte* buf, int size)
{
    int c = -1,  run = 0;
    auto out = buf;
    for (size_t i = 0; i < size; i++)
    {
        buf[i]==c?  run++  :  (run=1, c = *out++ = buf[i]);
    }
    return out-buf;
}

int main (int argc, char **argv)
{
    bool display_gpu = true;
    bool apply_lzp = true;
    bool apply_bwt = true;
    bool apply_rle = true;
    bool apply_mtf = true;
    int  mtf_num = -1;
    size_t bufsize = DEFAULT_BUFSIZE;
    char *comment;
    int error = 0;

    auto src_argv = argv,  dst_argv = argv;
    while (*++src_argv) {
      ParseBool (*src_argv, "-gpu", "-nogpu", &display_gpu) ||
      ParseBool (*src_argv, "-lzp", "-nolzp", &apply_lzp) ||
      ParseBool (*src_argv, "-bwt", "-nobwt", &apply_bwt) ||
      ParseBool (*src_argv, "-rle", "-norle", &apply_rle) ||
      ParseBool (*src_argv, "-mtf", "-nomtf", &apply_mtf) ||
      ParseInt  (*src_argv, "-mtf",           &mtf_num) ||
      ParseInt  (*src_argv, "-b",             &bufsize) ||
      ParseStr  (*src_argv, "-rem",           &comment) ||
      UnknownOption (*src_argv, &error) ||
      (*++dst_argv = *src_argv);
    }
    *++dst_argv = 0;  argc = dst_argv - argv;

    if (bufsize < 100*1000)
        bufsize <<= 20;  // megabytes

    if (!(argc==2 || argc==3) || error) {
        printf ("Usage: mtf [options] infile [outfile]\n"
                "  -nogpu   skip GPU name output\n"
                "  -nolzp   skip LZP transformation\n"
                "  -nobwt   skip BWT transformation\n"
                "  -norle   skip RLE transformation\n"
                "  -nomtf   skip MTF transformation\n"
                "  -mtfN    perform only MTF transformation number N\n"
                "  -bN      buffer N (mega)bytes\n"
                );
        return argc==1 && !error?  0 : 1;
    }

    unsigned char* d_inbuf;
    unsigned char* d_outbuf;
    checkCudaErrors( hipMalloc((void**)(&d_inbuf),  bufsize+CHUNK*2+256));  // up to CHUNK*2 extra bytes may be processed, plus a few extra bytes may be read after bufend
    checkCudaErrors( hipMalloc((void**)(&d_outbuf), bufsize+CHUNK*2+256));

    hipEvent_t start, stop;
    checkCudaErrors( hipEventCreate(&start));
    checkCudaErrors( hipEventCreate(&stop));

    unsigned char* inbuf  = new unsigned char[bufsize];
    unsigned char* outbuf = new unsigned char[bufsize];
    int*      bwt_tempbuf = apply_bwt? new int[bufsize] : 0;

    double insize = 0,  after_lzp = 0,  outsize = 0,  lzp_duration[2] = {0},  duration[100] = {0};  char *mtf_name[100] = {"cpu (1 thread)"};

    FILE* infile  = fopen (argv[1], "rb");
    FILE* outfile = fopen (argv[2]? argv[2] : "nul", "wb");
    if (!infile) {
        printf ("Can't open infile %s\n", argv[1]);
        return 2;
    }
    if (!outfile) {
        printf ("Can't open outfile %s\n", argv[3]);
        return 3;
    }
    if (display_gpu)
        DisplayCudaDevice();


    for (int inbytes; !!(inbytes = fread(inbuf,1,bufsize,infile)); )
    {
        insize += inbytes;
        byte *ptr = inbuf;  size_t outbytes = inbytes;  // output buffer

        if (apply_lzp) {
            lzp_cpu(inbuf, inbuf+inbytes, outbuf, outbuf+inbytes, 15, 32);

            StartTimer();
            lzp_cpu(inbuf, inbuf+inbytes, outbuf, outbuf+inbytes, 15, 32);
            lzp_duration[1] += GetTimer();

            StartTimer();
            auto lzp_errcode  =  bsc_lzp_encode_block(inbuf, inbuf+inbytes, outbuf, outbuf+inbytes, 15, 32);
            lzp_duration[0] += GetTimer();

            if (lzp_errcode != LIBBSC_NOT_COMPRESSIBLE) {
                if (lzp_errcode < 0) {
                    printf ("LZP failed with errcode %d\n", lzp_errcode);
                    return 4;
                }
                memcpy (inbuf, outbuf, inbytes=lzp_errcode);
            }
        }
        after_lzp += inbytes;

        if (apply_bwt) {
            auto bwt_errcode  =  sais_bwt (inbuf, outbuf, bwt_tempbuf, inbytes);
            if (bwt_errcode < 0) {
                printf ("BWT failed with errcode %d\n", bwt_errcode);
                return 5;
            }
            memcpy (inbuf, outbuf, inbytes);
        }

        if (0 == mtf_num  ||  mtf_num < 0) {
            StartTimer();
                unsigned char MTFTable[ALPHABET_SIZE];
                ptr = qlfc (inbuf, outbuf, inbytes, MTFTable);
                outbytes = outbuf+inbytes - ptr;
            duration[0] += GetTimer();
        }
        int num = 1;

        if (apply_rle) {
            inbytes = rle(inbuf,inbytes);
        }

        checkCudaErrors( hipMemcpy (d_inbuf, inbuf, inbytes, hipMemcpyHostToDevice));
        checkCudaErrors( hipDeviceSynchronize());

        auto time_run = [&] (char *name, std::function<void(void)> f) {
            mtf_name[num] = name;
            if (num == mtf_num  ||  mtf_num < 0)
            {
                checkCudaErrors( hipEventRecord (start, nullptr));
                f();
                checkCudaErrors( hipEventRecord (stop, nullptr));
                checkCudaErrors( hipDeviceSynchronize());

                if (num == mtf_num) {
                    checkCudaErrors( hipMemcpy (outbuf, d_outbuf, inbytes, hipMemcpyDeviceToHost));
                    checkCudaErrors( hipDeviceSynchronize());
                    ptr = outbuf;
                    outbytes = inbytes;
                }

                float start_stop;
                checkCudaErrors( hipEventElapsedTime (&start_stop, start, stop));
                duration[num] += start_stop;
            }
            num++;
        };

{
        const int NUM_WARPS = 4;
        time_run ("mtf_scalar        ", [&] {mtf_scalar    <CHUNK,NUM_WARPS>       <<<(inbytes-1)/(CHUNK*NUM_WARPS)+1,   NUM_WARPS*WARP_SIZE>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_2symbols      ", [&] {mtf_2symbols  <CHUNK,NUM_WARPS>       <<<(inbytes-1)/(CHUNK*NUM_WARPS)+1,   NUM_WARPS*WARP_SIZE>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_2buffers      ", [&] {mtf_2buffers  <CHUNK,NUM_WARPS>       <<<(inbytes-1)/(CHUNK*NUM_WARPS*2)+1, NUM_WARPS*WARP_SIZE>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_2buffers<32>  ", [&] {mtf_2buffers_depth32 <CHUNK,NUM_WARPS><<<(inbytes-1)/(CHUNK*NUM_WARPS*2)+1, NUM_WARPS*WARP_SIZE>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_3buffers<32>  ", [&] {mtf_2buffers_depth32 <CHUNK,NUM_WARPS,3><<<(inbytes-1)/(CHUNK*NUM_WARPS*3)+1, NUM_WARPS*WARP_SIZE>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_4buffers<32>  ", [&] {mtf_2buffers_depth32 <CHUNK,NUM_WARPS,4><<<(inbytes-1)/(CHUNK*NUM_WARPS*4)+1, NUM_WARPS*WARP_SIZE>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
}
{
        const int NUM_THREADS = 8*WARP_SIZE,  NUM_BUFFERS = NUM_THREADS/8;
        time_run ("mtf_4by8          ", [&]{mtf_4by8<CHUNK,NUM_THREADS,NUM_BUFFERS><<<(inbytes-1)/(CHUNK*NUM_BUFFERS)+1,         NUM_THREADS>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
}
{
        const int NUM_THREADS = 4*WARP_SIZE,  NUM_BUFFERS = NUM_THREADS/4;
        time_run ("mtf_8by4          ", [&]{mtf_4by8<CHUNK,NUM_THREADS,NUM_BUFFERS><<<(inbytes-1)/(CHUNK*NUM_BUFFERS)+1,         NUM_THREADS>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
}
        time_run ("mtf_thread        ", [&] {mtf_thread    <CHUNK>                 <<<(inbytes-1)/(CHUNK*WARP_SIZE)+1,             WARP_SIZE>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_thread_by4    ", [&] {mtf_thread_by4<CHUNK>                 <<<(inbytes-1)/(CHUNK*WARP_SIZE)+1,             WARP_SIZE>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});

        const int NUM_THREADS = 1*WARP_SIZE;
        time_run ("mtf_thread<8>     ", [&] {mtf_thread    <CHUNK,NUM_THREADS,8>   <<<(inbytes-1)/(CHUNK*NUM_THREADS)+1,         NUM_THREADS>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_thread<16>    ", [&] {mtf_thread    <CHUNK,NUM_THREADS,16>  <<<(inbytes-1)/(CHUNK*NUM_THREADS)+1,         NUM_THREADS>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_thread<32>    ", [&] {mtf_thread    <CHUNK,NUM_THREADS,32>  <<<(inbytes-1)/(CHUNK*NUM_THREADS)+1,         NUM_THREADS>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_thread<64>    ", [&] {mtf_thread    <CHUNK,NUM_THREADS,64>  <<<(inbytes-1)/(CHUNK*NUM_THREADS)+1,         NUM_THREADS>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});

        time_run ("mtf_thread_by4<8> ", [&] {mtf_thread_by4<CHUNK,NUM_THREADS,8>   <<<(inbytes-1)/(CHUNK*NUM_THREADS)+1,         NUM_THREADS>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_thread_by4<16>", [&] {mtf_thread_by4<CHUNK,NUM_THREADS,16>  <<<(inbytes-1)/(CHUNK*NUM_THREADS)+1,         NUM_THREADS>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_thread_by4<32>", [&] {mtf_thread_by4<CHUNK,NUM_THREADS,32>  <<<(inbytes-1)/(CHUNK*NUM_THREADS)+1,         NUM_THREADS>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});
        time_run ("mtf_thread_by4<64>", [&] {mtf_thread_by4<CHUNK,NUM_THREADS,64>  <<<(inbytes-1)/(CHUNK*NUM_THREADS)+1,         NUM_THREADS>>> (d_inbuf, d_outbuf, inbytes, CHUNK);});

        fwrite (ptr, 1, outbytes, outfile);
        outsize += outbytes;
    }

    auto print_stage_stats = [&] (char *name, double insize, double outsize, double duration) {
        printf("%s: %.0lf => %.0lf (%.2lf%%)", name, insize, outsize, outsize*100/insize);
        if (duration)
            printf("%5.0lf MiB/s,  %.3lf ms",  ((1000/duration) *  insize) / (1 << 20),  duration);
        printf("\n");
    };

    if (apply_lzp)  print_stage_stats ("lzp-bsc", insize, after_lzp, lzp_duration[0]);
    if (apply_lzp)  print_stage_stats ("lzp-cpu", insize, after_lzp, lzp_duration[1]);
    if (apply_rle)  print_stage_stats ("rle", after_lzp, outsize, 0);
    for (int i=0; i<sizeof(duration)/sizeof(*duration); i++) {
        if (duration[i]) {
            char in_speed[100], out_speed[100];
            sprintf( in_speed,   "%5.0lf", ((1000/duration[i]) *  insize) / (1 << 20));
            sprintf(out_speed, " /%5.0lf", ((1000/duration[i]) * outsize) / (1 << 20));
            printf("[%2d] %-*s: %s%s MiB/s,  %.3lf ms\n", i, strlen(mtf_name[2]), mtf_name[i], in_speed, (apply_rle?out_speed:""), duration[i]);
        }
    }
    fclose(infile);
    fclose(outfile);
    hipProfilerStop();
    return 0;
}
