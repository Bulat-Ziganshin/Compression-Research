#include "hip/hip_runtime.h"
// Copyright (C) 2016 Bulat Ziganshin
// All right reserved
// Part of https://github.com/Bulat-Ziganshin/Compression-Research

template <int CHUNK,  int NUM_THREADS = WARP_SIZE,  int MTF_SYMBOLS = ALPHABET_SIZE>
__global__ void mtf_thread_by4 (const byte* inbuf,  byte* outbuf,  int inbytes,  int chunk)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int tid = threadIdx.x;

    if (idx*CHUNK >= inbytes)  return;
    inbuf  += idx*CHUNK;
    outbuf += idx*CHUNK;
    auto cur  = *inbuf++;
    auto next = *inbuf++;

    volatile __shared__  byte mtf0 [MTF_SYMBOLS*NUM_THREADS];
    auto mtf = mtf0 + 4*tid;
    for (int k=0; k<MTF_SYMBOLS; k+=4)
    {
        *(unsigned*)(mtf+k*NUM_THREADS)  =  k + ((k+1)<<8) + ((k+2)<<16) + ((k+3)<<24);
    }


    int i = 0,  k = 0;
    auto mtf_k = mtf;
    auto old = cur;

    for(;;)
    {
        #pragma unroll
        for (int x=0; x<4; x++)
        {
            auto next = mtf_k[x];
            mtf_k[x] = old;
            old = next;
            if (cur==old)  goto found;
            k++;
        }
        mtf_k += 4*NUM_THREADS;
        if (MTF_SYMBOLS == ALPHABET_SIZE  ||  k < MTF_SYMBOLS)
            continue;

found:
        *outbuf++ = k;
        if (++i >= CHUNK)  return;

        old = cur = next;
        next = *inbuf++;

        mtf_k = mtf;
        k = 0;
    }
}
