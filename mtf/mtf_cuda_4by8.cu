#include "hip/hip_runtime.h"
// Copyright (c) 2016 Bulat Ziganshin <Bulat.Ziganshin@gmail.com>
// All rights reserved
// Part of https://github.com/Bulat-Ziganshin/Compression-Research

template <int CHUNK,  int NUM_THREADS,  int NUM_BUFFERS,  typename MTF_WORD = byte,  int MTF_SYMBOLS = ALPHABET_SIZE>
__global__ void mtf_cuda_4by8 (const byte* __restrict__ inbuf,  byte* __restrict__ outbuf,  int inbytes,  int chunk)
{
    // NUM_BUFFERS   - how many buffers processed by each thread block
    // NUM_POSITIONS - how many positions in each buffer are checked simultaneously (i.e. by the single warp)
    const int NUM_POSITIONS  =  NUM_THREADS / NUM_BUFFERS;
    const int idx = (blockIdx.x * blockDim.x + threadIdx.x) / NUM_POSITIONS;
    const int buf = threadIdx.x / NUM_POSITIONS;
    const int pos = threadIdx.x % NUM_POSITIONS;
    const int first_bit = (buf * NUM_POSITIONS) % WARP_SIZE;

    if (idx*CHUNK >= inbytes)  return;
    inbuf  += idx*CHUNK;
    outbuf += idx*CHUNK;
    auto cur  = *inbuf++;
    auto next = *inbuf++;

    volatile __shared__  MTF_WORD mtf0 [MTF_SYMBOLS*NUM_BUFFERS];
    auto mtf = mtf0 + buf*MTF_SYMBOLS;
    auto mtf_pos = mtf+pos;
    for (int k=0; k<MTF_SYMBOLS; k+=NUM_POSITIONS)
    {
        mtf_pos[k] = k+pos;
    }
    //__syncthreads();


    int i = 0,  k = 0;
    auto old  = *mtf_pos;

    for(;;)
    {
        unsigned n = __ballot (cur==old);                       // combined flags for NUM_POSITIONS in NUM_BUFFERS
        if (NUM_POSITIONS < WARP_SIZE)
            n  =  (n >> first_bit) % (1<<NUM_POSITIONS);        // only NUM_POSITIONS flags for the current buffer
        if (n==0) {                                             // if there is no match among these positions in the current buffer
            auto next = mtf_pos[k+NUM_POSITIONS];
            mtf_pos[k+1] = old;
            old = next;
            k += NUM_POSITIONS;
            //__syncthreads();
        } else {
            auto minbit = __ffs(n) - 1;
            if (pos < minbit)  mtf_pos[k+1] = old;
            *outbuf++ = k+minbit;
            mtf[0] = cur;
            //__syncthreads();
            old = *mtf_pos;
            if (++i >= CHUNK)  return;

            cur = next;
            next = *inbuf++;
            k = 0;
        }
    }
}
